#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "Sensor.h"
#include <stdio.h>

#define NUM_SENSORS 50

__global__ void readSensorVals(double *readings, Sensor *sensors)
{
    int i = threadIdx.x;
    if(i < NUM_SENSORS)
    readings[i] = sensors[i].read();
}

int main()
{
    double readings[NUM_SENSORS];
    Sensor sensors[NUM_SENSORS];

    hipMalloc(&readings, NUM_SENSORS * sizeof(double));
    hipMalloc(&sensors, NUM_SENSORS * sizeof(Sensor));

    readSensorVals <<<1, NUM_SENSORS >>> (&readings,&sensors);

    hipFree(readings);
    hipFree(sensors);

}
